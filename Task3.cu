//importing all the header files

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


//compile with nvcc Task3.cu -o Task3
//./Task3 Passpassword twouppercaseandtwodigits
//__global__ --> GPU function which can be launched by many blocks and threads, defination of kernel which is launched on the GPU whenever called from the CPU
//__device__ --> GPU function or variables
//__host__ --> CPU function or variables

//Function called on GPU executed in GPU for password encryption on the cmd argument
__device__ char* CudaCrypt(char* Plainpassword){

       //typecasting for memory allocation on GPU for encrypt password
	char * Freshpassword = (char *) malloc(sizeof(char) * 11);
	
        //limiting the encrypt password to display till 10th index and terminating it after when it reaches the 11th index
	Freshpassword[0] = Plainpassword[0] + 2;  
	Freshpassword[1] = Plainpassword[0] - 2;
	Freshpassword[2] = Plainpassword[0] + 1;  
	Freshpassword[3] = Plainpassword[1] + 3;
	Freshpassword[4] = Plainpassword[1] - 3;
	Freshpassword[5] = Plainpassword[1] - 1;
	Freshpassword[6] = Plainpassword[2] + 2;
	Freshpassword[7] = Plainpassword[2] - 2;
	Freshpassword[8] = Plainpassword[3] + 4;
	Freshpassword[9] = Plainpassword[3] - 4;
	Freshpassword[10] = '\0';
//loop for checking password character by character and number by number
	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all upper case letter limits
		//ASCII value of A-Z = 65-90
		//ASCII value of 0-9 = 48-57
			if(Freshpassword[i] > 90){
				Freshpassword[i] = (Freshpassword[i] - 90) + 65;
			}else if(Freshpassword[i] < 65){
				Freshpassword[i] = (65 - Freshpassword[i]) + 65;
			}
		}else{ //checking number limits
			if(Freshpassword[i] > 57){
				Freshpassword[i] = (Freshpassword[i] - 57) + 48;
			}else if(Freshpassword[i] < 48){
				Freshpassword[i] = (48 - Freshpassword[i]) + 48;
			}
		}
	}
	return Freshpassword; //encrypted password is returned
}
//function for comparing two strings runs on GPU which stores value of the password and stores in encPassword
__device__ int compareTwoString(char* StringOne, char* StringTwo){
	
    while(*StringOne)
    {
        //two strings being compared
        if (*StringOne != *StringTwo)
            break;
 
        //Changing Pointer location
        StringOne++;
        StringTwo++;
    }
 
    // if the two strings matches it returns 0 
    return *(const unsigned char*)StringOne - *(const unsigned char*)StringTwo;
}

//function called on the CPU which is executed on the GPU here the user given password is hashed
__global__ void crack(char * Alpha, char * Num, char * Plainpassword){

char genFreshPass[4];
//Adding test passwords to genFreshPass
genFreshPass[0] = Alpha[blockIdx.x];
genFreshPass[1] = Alpha[blockIdx.y];

genFreshPass[2] = Num[threadIdx.x];
genFreshPass[3] = Num[threadIdx.y];

//Plain Password being encrypted
char *encPassword = CudaCrypt(Plainpassword);
	
	//Comparing encrypted genFreshPass with encPassword
	if(compareTwoString(CudaCrypt(genFreshPass),encPassword) == 0){
		printf("Your to be cracked password is : %s = %s\n",genFreshPass, Plainpassword);
		printf("Your cracked password is : %s = %s\n", encPassword);

	}
}

int time_difference(struct timespec *initial, struct timespec *final, long long int *Diff){
  long long int ds =  final->tv_sec - initial->tv_sec; 
  long long int dn =  final->tv_nsec - initial->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *Diff = ds * 1000000000 + dn;
  return !(*Diff > 0);
}


//Main function which is executed on the CPU
int main(int argc, char ** argv){
//storing all the 26 alphabets in the variable cpuAlpha
char CPUAlpha[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
//storing the 10 numbers in the variable name cpuNum
char CPUNum[10] = {'0','1','2','3','4','5','6','7','8','9'};

char * GPUAlpha;
// allocating GPU memory
hipMalloc( (void**) &GPUAlpha, sizeof(char) * 26); 
// copy back the result array of the alphabeth to the CPU
hipMemcpy(GPUAlpha, CPUAlpha, sizeof(char) * 26, hipMemcpyHostToDevice);

char * GPUNum;
// allocating GPU memory
hipMalloc( (void**) &GPUNum, sizeof(char) * 10); 
// copy back the result array of the number to the CPU
hipMemcpy(GPUNum, CPUNum, sizeof(char) * 10, hipMemcpyHostToDevice);

char * PASSW;
// allocating GPU memory
hipMalloc( (void**) &PASSW, sizeof(char) * 26); 
// copy back the result array of the result password to the CPU
hipMemcpy(PASSW, argv[1], sizeof(char) * 26, hipMemcpyHostToDevice);

	struct timespec initial, final;
	long long int time_taken;
	
//Starting to monitor the time duration 
	clock_gettime(CLOCK_MONOTONIC, &initial);
	// launching the kernel
	crack<<< dim3(26,26,1), dim3(10,10,1) >>>( GPUAlpha, GPUNum, PASSW);
	hipDeviceSynchronize();

//Ending the duration of the program
	clock_gettime(CLOCK_MONOTONIC, &final);
	
//Calculating the duration of the time for exection of the program
	time_difference(&initial, &final, &time_taken);
	
//Printing the duration taken for execution of the program
	printf("Time taken was %lldns or %0.9lfs\n", time_taken,
                                         (time_taken/1.0e9));

return 0;
}


	


